﻿#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    if (n == 0) return {};
    std::vector<float> c(n * n);
    size_t size = n * n * sizeof(float);

    float *d_a, *d_b, *d_c, *d_ct;

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);
    hipMalloc(&d_ct, size);

    hipblasSetMatrix(n, n, sizeof(float), a.data(), n, d_a, n);
    hipblasSetMatrix(n, n, sizeof(float), b.data(), n, d_b, n);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, n, n, &alpha, d_a, n, d_b, n, &beta, d_c, n);
    hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, &alpha, d_с, n, &beta, nullptr, n, d_сt, n);

    hipblasGetMatrix(n, n, sizeof(float), d_ct, n, c.data(), n);

    hipblasDestroy(handle);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_ct);

    return c;
}